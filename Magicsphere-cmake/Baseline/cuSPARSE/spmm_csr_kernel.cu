#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <torch/extension.h>


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int cuSPARSE_spmm_csr_kernel(int* dA_csrOffsets, 
                             int* dA_columns, 
                             float* dA_values, 
                             float* dB, 
                             float *dC,
                             const long dimM, 
                             const long dimN, 
                             const long nnz)
{
    const long ldb = dimN; 
    const long ldc = dimN; 
    float alpha = 1.0f;
    float beta = 0.0f;
        //     printf("%d ",*(dA_csrOffsets));
        // printf("\n");

    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )

    //以CSR格式创建A矩阵
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, dimM, dimM, nnz,
                    dA_csrOffsets, dA_columns, dA_values,
                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )

    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, dimM, dimN, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )

    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, dimM, dimN, ldc, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )

    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(handle,
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                            HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    CHECK_CUDA( hipFree(dBuffer) )
    return EXIT_SUCCESS;
}

torch::Tensor cuSPARSE_spmm_csr(
    torch::Tensor row_offsets,
    torch::Tensor col_indices, 
    torch::Tensor values, 
    torch::Tensor rhs_matrix,
    const long dimM,
    const long dimN,
    const long nnz){

    auto output_matrix = torch::zeros({dimM,dimN}, torch::kCUDA);

    cuSPARSE_spmm_csr_kernel(
        row_offsets.data<int>(),
        col_indices.data<int>(),
        values.data<float>(),
        rhs_matrix.data<float>(),
        output_matrix.data<float>(),
        dimM,
        dimN,
        nnz
    );

    return output_matrix;
}




// sddmm
int cuSPARSE_sddmm_csr_kernel(int * dC_csrOffsets, 
                             int * dC_columns, 
                             float * dA, 
                             float * dB, 
                             float *dC,
                             const long dimM, 
                             const long dimN, 
                             const long nnz)
{
    const long lda = dimN; 
    const long ldb = dimN; 
    float alpha = 1.0f;
    float beta = 0.0f;

    hipsparseHandle_t     handle = NULL;
    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )

    // Create dense matrix A
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, dimM, dimN, lda, dA,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )

    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, dimN, dimM, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )

    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, dimM, dimM, nnz,
                                      dC_csrOffsets, dC_columns, dC,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )

    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSDDMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize) )

    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSDDMM(
                                  handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                  HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    CHECK_CUDA( hipFree(dBuffer) )
    return EXIT_SUCCESS;
}

torch::Tensor cuSPARSE_sddmm_csr(
    torch::Tensor row_offsets,
    torch::Tensor col_indices, 
    torch::Tensor lhs_matrix,
    torch::Tensor rhs_matrix,
    const long dimM,
    const long dimN,
    const long nnz){

    auto output_matrix = torch::zeros({nnz}, torch::kCUDA).to(torch::kF32);

    cuSPARSE_sddmm_csr_kernel(
        row_offsets.data<int>(),
        col_indices.data<int>(),
        lhs_matrix.data<float>(),
        rhs_matrix.data<float>(),
        output_matrix.data<float>(),
        dimM,
        dimN,
        nnz
    );

    return output_matrix;
}