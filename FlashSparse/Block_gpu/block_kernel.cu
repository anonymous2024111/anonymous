#include "hip/hip_runtime.h"
#include "config.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <fstream>
#include <mma.h>
// #include <sputnik/spmm/cuda_spmm.h>
// #include <sputnik/sputnik.h>
#include <sstream>
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <torch/extension.h>
#include <vector>
#define WPB 8
#define EXE_TIME 10
#define NUM_SM_GPU 128 // 4090
#define USE_SPUTNIK
using namespace nvcuda;

struct GpuTimer {
  hipEvent_t start;
  hipEvent_t stop;
  GpuTimer() {
    hipEventCreate(&start);
    hipEventCreate(&stop);
  }

  ~GpuTimer() {
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  void Start() { hipEventRecord(start); }

  void Stop() { hipEventRecord(stop); }

  float Elapsed() {
    float elapsed;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    return elapsed;
  }
};

// From (https://github.com/xxcclong/GNN-Computing)
typedef uint64_t clocktype;
struct Dur {
  clocktype begin;
  clocktype end;
  int smid = -1;
  Dur(clocktype x, clocktype y, int outsm) {
    begin = x;
    end = y;
    smid = outsm;
  }
};

bool cmp(Dur x, Dur y) { return (x.end > y.end); }
static __device__ inline uint64_t GlobalTimer64(void) {
  volatile uint64_t first_reading;
  volatile uint32_t second_reading;
  uint32_t high_bits_first;
  asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(first_reading));
  high_bits_first = first_reading >> 32;
  asm volatile("mov.u32 %0, %%globaltimer_hi;" : "=r"(second_reading));
  if (high_bits_first == second_reading) {
    return first_reading;
  }
  // Return the value with the updated high bits, but the low bits set to 0.
  return ((uint64_t)second_reading) << 32;
}
__device__ inline uint getSMId() {
  uint smid;
  asm("mov.u32 %0, %smid;" : "=r"(smid));
  return smid;
}

//////////////////////////////////////////////////////////////////////
/// Preprocessing
//////////////////////////////////////////////////////////////////////
__global__ void roundup_to_multiple_of_eight(int *input, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size) {
    int rounded_value = ((input[tid] + 7) / 8) * 8;
    input[tid] = rounded_value;
  }
}

__global__ void get_padding_tileid_kernel(int *ori_offset, uint8_t *ori_tileid,
                                          int *padded_offset,
                                          uint8_t *padded_tileid, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < size) {
    int s = ori_offset[tid];
    int e = ori_offset[tid + 1];
    int s1 = padded_offset[tid];
    for (int i = 0; i < e - s; i++) {
      padded_tileid[s1 + i] = ori_tileid[s + i];
    }
  }
}


__global__ void fill_edgeToRow(int *edgeToRow, int *nodePointer,
                               int num_nodes) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int nid = tid / 32;
  int laneid = tid % 32;
  // check a valid node range.
  if (nid < num_nodes) {
#pragma unroll
    for (int eid = nodePointer[nid] + laneid; eid < nodePointer[nid + 1];
         eid += 32) {
      edgeToRow[eid] = nid;
    }
  }
}

void fill_edgeToRow_cuda(int *edgeToRow, int *nodePointer, int num_nodes) {
  int wrap_size = 32;
  int block_size = 1024;
  int grid_size = (num_nodes * wrap_size + block_size - 1) / block_size;
  fill_edgeToRow<<<grid_size, block_size>>>(edgeToRow, nodePointer, num_nodes);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}

/*Generate segment*/
__global__ void fill_segment(int *nodePointer, int *seg_out, int blockSize_h,
                             int blockSize_w, int num_nodes) {
  int tid = threadIdx.x;
  int winId = blockIdx.x; // each block one window
  //Window开始的行
  unsigned block_start = nodePointer[winId * blockSize_h];
  //Window结束的行
  unsigned block_end =
      nodePointer[min(winId * blockSize_h + blockSize_h, num_nodes)];
  //window内非零元个数
  unsigned num_window_edges = block_end - block_start;
//   if(winId==0 && threadIdx.x==0){
// 	printf("%d\n", num_window_edges);
//   }
  const unsigned threadPerBlock = blockDim.x * blockDim.y;
  for (unsigned idx = tid; idx < num_window_edges; idx += threadPerBlock) {
    seg_out[block_start + idx] = winId;
  }
}

void fill_segment_cuda(int *nodePointer, int *seg_out, int blockSize_h,
                       int blockSize_w, int num_nodes) {
  // 每个window由512个线程负责
  int block_size = 512;
  int window_count = (num_nodes + blockSize_h - 1) / blockSize_h;
  fill_segment<<<window_count, block_size>>>(nodePointer, seg_out, blockSize_h,
                                             blockSize_w, num_nodes);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}

/*Generate TCblock_rowid*/
__global__ void generate_tcblock_rowid(int *rowwindow_offset,
                                       int *tcblock_rowid,
                                       int num_row_windows) {
  int tid = threadIdx.x;
  int winId = blockIdx.x; // each warp one window
  unsigned block_start = rowwindow_offset[winId];
  unsigned block_end = rowwindow_offset[min(winId + 1, num_row_windows)];
  unsigned num_blocks = block_end - block_start;
  const unsigned threadPerBlock = blockDim.x * blockDim.y;
  for (unsigned idx = tid; idx < num_blocks; idx += threadPerBlock) {
    tcblock_rowid[block_start + idx] = winId;
  }
}
void generate_tcblock_rowid_cuda(int *rowwindow_offset, int *tcblock_rowid,
                                 int num_row_windows) {
  int block_size = 512;
  int window_count = num_row_windows;
  generate_tcblock_rowid<<<window_count, block_size>>>(
      rowwindow_offset, tcblock_rowid, num_row_windows);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}

/* Generate edge2column*/
__device__ __forceinline__ int binarysearch(int *arr, int size, int target) {
  int left = 0;
  int right = size - 1;
  while (left <= right) {
    int mid = left + (right - left) / 2;
    if (arr[mid] == target) {
      while (mid > 0 && arr[mid - 1] == target) {
        mid--;
      }
      return mid;
    } else if (arr[mid] < target) {
      left = mid + 1;
    } else {
      right = mid - 1;
    }
  }
  return -1;
}
__device__ __forceinline__ void inplace_deduplication(int *array, int length,
                                                      int *loc) {
  int cur = 1;
  while (cur < length) {
    if (array[cur] != array[cur - 1]) {
      (*loc)++;
      array[(*loc)] = array[cur];
    }
    cur++;
  }

  (*loc)++;
}

__device__ __forceinline__ void inplace_deduplication_libra_spmm(int *array, int *counts, int length, int *loc) {
  int count = 1; // 记录当前元素的计数
  for (int cur = 1; cur < length; cur++) {
    if (array[cur] != array[cur - 1]) {
      counts[*loc] = count; // 保存上一个元素的计数
      (*loc)++;             // 更新位置
      array[*loc] = array[cur]; // 将当前元素写入去重数组
      count = 1;            // 重置计数器
    } else {
      count++; // 若相同则增加当前元素的计数
    }
  }

  counts[*loc] = count; // 保存最后一个元素的计数
  (*loc)++;             // 更新位置，表示最终去重后元素个数
}

//去重，以及求vector_num
__device__ __forceinline__ void distribute_libra_spmm(int *array, int *counts, int length, int *loc,
			int threshold, int *vector_num, int *vector_nnz) {
	int count = 1; // 记录当前元素的计数
	for (int cur = 1; cur < length; cur++) {
		if (array[cur] != array[cur - 1]) {
			counts[*loc] = count; // 保存上一个元素的计数
			(*loc)++;             // 更新位置
			array[*loc] = array[cur]; // 将当前元素写入去重数组
			//判断是否超过阈值
			if(count>=threshold){
				(*vector_num)++;
				(*vector_nnz)+=count;
			}
			count = 1;            // 重置计数器
		} else {
			count++; // 若相同则增加当前元素的计数
		}
	}
	counts[*loc] = count; // 保存最后一个元素的计数
	(*loc)++;             // 更新位置，表示最终去重后元素个数
}

__device__ __forceinline__ void distribute_cuda_tile_libra_spmm(
	int *counts_cur, int *edgetocol, int start_row, int num_nodes,
	int *nodePointer, int threshold, int Short_len, int c_s, int *cuda_long, int* cuda_short,
	int *cuda_long_group, int * cuda_short_group) {

	int cur = 0;
	//遍历每一行,统计每行cuda tile的元素个数
	for (int cur_row = start_row; cur_row < min(start_row+8, num_nodes); cur_row++) {
		//遍历当前行的所有元素
		for(int m=nodePointer[cur_row]; m<nodePointer[cur_row+1]; m++){
			//如果当前元素的newcol的值小于threshold,则交由CUDA tile
			int col_density = counts_cur[edgetocol[m]];
			if(col_density < threshold){
				cuda_long[cur]++;
			}
		}
		cur++;
	}

	//拆分cuda_long
	for(int i=0; i<8; i++){
		//如果是短行
		if(cuda_long[i]<= Short_len)
		{
			cuda_short[i] = cuda_long[i];
			cuda_long[i] = 0;
			(*cuda_short_group)++;
		}else{
			//如果是长行, 是否需要差分
			if(cuda_long[i]<=c_s){
				//不需要拆分
				(*cuda_long_group)++;
			}else{
				//需要拆分
				(*cuda_long_group) += cuda_long[i]/c_s;
				//判断residue是否存在
				int residue = (cuda_long[i]%c_s);
				if(residue> 0)
				{
					//residue是短行
					if(residue<= Short_len)
					{				
						cuda_short[i] = residue;
						cuda_long[i] -= residue;
						(*cuda_short_group)++;
					}else{
						(*cuda_short_group)++;
					}
				}
			}
		}
	}


}



__global__ void generate_edgetocolumn(int *nodePointer, int *edgelist,
                                      int *edgelist_sort, int *edgetocol,
                                      int *blockpartition, int *blocknum,
                                      int blockSize_h, int blockSize_w,
                                      int num_nodes) {
  int winId = blockIdx.x; // each warp one window
  unsigned block_start = nodePointer[winId * blockSize_h];
  unsigned block_end =
      nodePointer[min(winId * blockSize_h + blockSize_h, num_nodes)];
  unsigned num_window_edges = block_end - block_start;
  if (num_window_edges == 0)
    return;
  const unsigned threadPerBlock = blockDim.x * blockDim.y;
  int *start = edgelist_sort + block_start;
  int size = 0;
  //去重
  inplace_deduplication(start, num_window_edges, &size);
  //num是每个窗口有多少个block
  int num = (size + blockSize_w) / blockSize_w;
  atomicAdd(blocknum, num);
  blockpartition[winId] = num;
  for (unsigned idx = block_start; idx < block_end; idx += 1) {
    int index = binarysearch(start, size + 1, edgelist[idx]);
    edgetocol[idx] = index;
  }
}
void generate_edgetocolumn_cuda(int *nodePointer, int *edgelist,
                                int *edgelist_sort, int *edgetocol,
                                int *blockpartition, int *blocknum,
                                int blockSize_h, int blockSize_w,
                                int num_nodes) {
  int block_size = 1;
  int window_count = (num_nodes + blockSize_h - 1) / blockSize_h;
  //每个block负责一个window, 每个block中只有一个线程
//   int block_size1 = 128;
//   int block_count1 = (window_count + 127) / 128;
  generate_edgetocolumn<<<window_count, block_size>>>(
      nodePointer, edgelist, edgelist_sort, edgetocol, blockpartition, blocknum,
      blockSize_h, blockSize_w, num_nodes);
  // generate_edgetocolumn_v1<<< window_count, block_size >>> (nodePointer,
  // edgelist, edgelist_sort, edgetocol, blockpartition, blocknum, blockSize_h,
  // blockSize_w, num_nodes);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}

__global__ void generate_edgetocolumn_fs(int *nodePointer, int *edgelist,
                                      int *edgelist_sort, int *edgetocol,
                                      int *blockpartition, int *blocknum, int *vectornum,
                                      int blockSize_h, int blockSize_w,
                                      int num_nodes) {
  int winId = blockIdx.x; // each warp one window
  unsigned block_start = nodePointer[winId * blockSize_h];
  unsigned block_end =
      nodePointer[min(winId * blockSize_h + blockSize_h, num_nodes)];
  unsigned num_window_edges = block_end - block_start;
  if (num_window_edges == 0)
    return;
  const unsigned threadPerBlock = blockDim.x * blockDim.y;
  int *start = edgelist_sort + block_start;
  int size = 0;
  int num = 0;
  //去重
  inplace_deduplication(start, num_window_edges, &size);

  //num是每个窗口有多少个block
  if(size>0)
  num = (size + blockSize_w - 1) / blockSize_w;
  atomicAdd(blocknum, num);
  atomicAdd(vectornum, size);
  //vector个数
  blockpartition[winId] = size;
  for (unsigned idx = block_start; idx < block_end; idx += 1) {
    int index = binarysearch(start, size + 1, edgelist[idx]);
    edgetocol[idx] = index;
  }
}
void generate_edgetocolumn_cuda_fs(int *nodePointer, int *edgelist,
                                int *edgelist_sort, int *edgetocol,
                                int *blockpartition, int *blocknum, int * vectornum,
                                int blockSize_h, int blockSize_w,
                                int num_nodes) {
  int block_size = 1;
  int window_count = (num_nodes + blockSize_h - 1) / blockSize_h;
  //每个block负责一个window, 每个block中只有一个线程
//   int block_size1 = 128;
//   int block_count1 = (window_count + 127) / 128;
  generate_edgetocolumn_fs<<<window_count, block_size>>>(
      nodePointer, edgelist, edgelist_sort, edgetocol, blockpartition, blocknum, vectornum,
      blockSize_h, blockSize_w, num_nodes);
  // generate_edgetocolumn_v1<<< window_count, block_size >>> (nodePointer,
  // edgelist, edgelist_sort, edgetocol, blockpartition, blocknum, blockSize_h,
  // blockSize_w, num_nodes);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}


__global__ void generate_edgetocolumn_fs_ori(int *nodePointer, int *edgelist,
                                      int *edgelist_sort, int *edgetocol,
                                      int *blockpartition, int *blocknum, int *vectornum,
                                      int blockSize_h, int blockSize_w,
                                      int num_nodes) {
  int winId = blockIdx.x; // each warp one window
  unsigned block_start = nodePointer[winId * blockSize_h];
  unsigned block_end =
      nodePointer[min(winId * blockSize_h + blockSize_h, num_nodes)];
  unsigned num_window_edges = block_end - block_start;
  if (num_window_edges == 0)
    return;
  const unsigned threadPerBlock = blockDim.x * blockDim.y;
  int *start = edgelist_sort + block_start;
  int size = 0;
  int num = 0;
  //去重
  inplace_deduplication(start, num_window_edges, &size);

  //num是每个窗口有多少个block
  if(size>0)
  num = (size + blockSize_w - 1) / blockSize_w;
  atomicAdd(blocknum, num);
  atomicAdd(vectornum, num*blockSize_w);
  //vector个数
  blockpartition[winId] = num*blockSize_w;
  for (unsigned idx = block_start; idx < block_end; idx += 1) {
    int index = binarysearch(start, size + 1, edgelist[idx]);
    edgetocol[idx] = index;
  }
}
void generate_edgetocolumn_cuda_fs_ori(int *nodePointer, int *edgelist,
                                int *edgelist_sort, int *edgetocol,
                                int *blockpartition, int *blocknum, int * vectornum,
                                int blockSize_h, int blockSize_w,
                                int num_nodes) {
  int block_size = 1;
  int window_count = (num_nodes + blockSize_h - 1) / blockSize_h;
  generate_edgetocolumn_fs_ori<<<window_count, block_size>>>(
      nodePointer, edgelist, edgelist_sort, edgetocol, blockpartition, blocknum, vectornum,
      blockSize_h, blockSize_w, num_nodes);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}

__global__ void generate_edgetocolumn_fs_balance(int *nodePointer, int *edgelist,
                                      int *edgelist_sort, int *edgetocol,
                                      int *blockpartition, int *vectorpartition, int *blocknum, int *vectornum,
                                      int blockSize_h, int blockSize_w,
                                      int num_nodes, int part) {
  int winId = blockIdx.x; // each warp one window
  unsigned block_start = nodePointer[winId * blockSize_h];
  unsigned block_end =
      nodePointer[min(winId * blockSize_h + blockSize_h, num_nodes)];
  unsigned num_window_edges = block_end - block_start;
  if (num_window_edges == 0)
    return;
  const unsigned threadPerBlock = blockDim.x * blockDim.y;
  int *start = edgelist_sort + block_start;
  int size = 0;
  int num = 0;
  //去重
  inplace_deduplication(start, num_window_edges, &size);

  //num是每个窗口有多少个block
  if(size>0)
  num = (size + blockSize_w - 1) / blockSize_w;
  //group 个数
  int group = (num + part - 1) / part;
  atomicAdd(blocknum, group);
  atomicAdd(vectornum, size);
  blockpartition[winId] = group;
  vectorpartition[winId] = size;
  for (unsigned idx = block_start; idx < block_end; idx += 1) {
    int index = binarysearch(start, size + 1, edgelist[idx]);
    edgetocol[idx] = index;
  }
}
void generate_edgetocolumn_cuda_fs_balance(int *nodePointer, int *edgelist,
                                int *edgelist_sort, int *edgetocol,
                                int *blockpartition, int *vectorpartition, int *blocknum, int * vectornum,
                                int blockSize_h, int blockSize_w,
                                int num_nodes, int part) {
  int block_size = 1;
  int window_count = (num_nodes + blockSize_h - 1) / blockSize_h;
  generate_edgetocolumn_fs_balance<<<window_count, block_size>>>(
      nodePointer, edgelist, edgelist_sort, edgetocol, blockpartition, vectorpartition, blocknum, vectornum,
      blockSize_h, blockSize_w, num_nodes, part);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}

/*Generate TC offset, tileid and AtoB*/
__global__ void generate_tcoffset_id_atob_fs(
    int *nodePointer, int *rowwindow_offset, int *edgeToColumn, int *edgeToRow,
    int *edgeList, half *values,
    int *sparseatob, int max_block, int num_nodes, long blockSize_h,
    int blockSize_w, int num_row_windows) {
//   extern __shared__ int pos_ptr[];
  int tid = threadIdx.x;
  int winId = blockIdx.x; // each warp one window
  unsigned vector_start = rowwindow_offset[winId];
  unsigned vector_end = rowwindow_offset[min(winId + 1, num_row_windows)];
  unsigned num_vector = vector_end - vector_start;
  if (num_vector == 0) {
    return;
  }
  unsigned element_start = nodePointer[winId * blockSize_h];
  unsigned element_end =
      nodePointer[min(int(winId * blockSize_h + blockSize_h), num_nodes)];
  unsigned num_window_edges = element_end - element_start;
  if (num_window_edges == 0) {
    return;
  }
  //开始看每个非零元在block内的偏移了
//   auto tileid = tcblocktile_id + element_start;
  auto values_ = values + vector_start*blockSize_h;
  auto sparse_AToB = sparseatob + vector_start;
  for (unsigned e_index = element_start; e_index < element_end; e_index++) {
    unsigned col = edgeToColumn[e_index]; // new col
    unsigned tcblock_id = col / blockSize_w;
    unsigned row_local = edgeToRow[e_index] % blockSize_h;
    unsigned col_local = col % blockSize_w;

	//如果存在， 且元素在residue里，需要按每行residue偏移
	int residue = num_vector % blockSize_w;
	if(residue>0 & col>=(num_vector-residue)){
		values_[tcblock_id*blockSize_h*blockSize_w + row_local*residue + col_local] = __float2half(1.0);
	}else{
		values_[tcblock_id*blockSize_h*blockSize_w + row_local*blockSize_w + col_local] = __float2half(1.0);
	}
	sparse_AToB[tcblock_id * blockSize_w + col_local] = edgeList[e_index];
    // pos_ptr[tcblock_id]++;
  }
}

void generate_tcoffset_id_atob_cuda_fs(int *nodePointer, int *rowwindow_offset,
                                    int *edgeToColumn, int *edgeToRow,
                                    int *edgeList, half *values, int *sparseatob,
                                    int max_block, int num_nodes,
                                    int blockSize_h, int blockSize_w,
                                    int num_row_windows) {
  int block_size = 1;
  int window_count = num_row_windows;
  generate_tcoffset_id_atob_fs<<<window_count, block_size>>>(
      nodePointer, rowwindow_offset, edgeToColumn, edgeToRow, edgeList,
    values, sparseatob, max_block, num_nodes,
      blockSize_h, blockSize_w, num_row_windows);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}


__global__ void generate_tcoffset_id_atob_fs_ori(
    int *nodePointer, int *rowwindow_offset, int *edgeToColumn, int *edgeToRow,
    int *edgeList, half *values,
    int *sparseatob, int max_block, int num_nodes, long blockSize_h,
    int blockSize_w, int num_row_windows) {
//   extern __shared__ int pos_ptr[];
  int tid = threadIdx.x;
  int winId = blockIdx.x; // each warp one window
  unsigned vector_start = rowwindow_offset[winId];
  unsigned vector_end = rowwindow_offset[min(winId + 1, num_row_windows)];
  unsigned num_vector = vector_end - vector_start;
  if (num_vector == 0) {
    return;
  }
  unsigned element_start = nodePointer[winId * blockSize_h];
  unsigned element_end =
      nodePointer[min(int(winId * blockSize_h + blockSize_h), num_nodes)];
  unsigned num_window_edges = element_end - element_start;
  if (num_window_edges == 0) {
    return;
  }
  //开始看每个非零元在block内的偏移了
//   auto tileid = tcblocktile_id + element_start;
  auto values_ = values + vector_start*blockSize_h;
  auto sparse_AToB = sparseatob + vector_start;
  for (unsigned e_index = element_start; e_index < element_end; e_index++) {
    unsigned col = edgeToColumn[e_index]; // new col
    unsigned tcblock_id = col / blockSize_w;
    unsigned row_local = edgeToRow[e_index] % blockSize_h;
    unsigned col_local = col % blockSize_w;

	//如果存在， 且元素在residue里，需要按每行residue偏移
	values_[tcblock_id*blockSize_h*blockSize_w + row_local*blockSize_w + col_local] = __float2half(1.0);
	
	sparse_AToB[tcblock_id * blockSize_w + col_local] = edgeList[e_index];
    // pos_ptr[tcblock_id]++;
  }
}

void generate_tcoffset_id_atob_cuda_fs_ori(int *nodePointer, int *rowwindow_offset,
                                    int *edgeToColumn, int *edgeToRow,
                                    int *edgeList, half *values, int *sparseatob,
                                    int max_block, int num_nodes,
                                    int blockSize_h, int blockSize_w,
                                    int num_row_windows) {
  int block_size = 1;
  int window_count = num_row_windows;
  generate_tcoffset_id_atob_fs_ori<<<window_count, block_size>>>(
      nodePointer, rowwindow_offset, edgeToColumn, edgeToRow, edgeList,
    values, sparseatob, max_block, num_nodes,
      blockSize_h, blockSize_w, num_row_windows);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}


__global__ void generate_tcoffset_id_atob_fs_balance(
    int *nodePointer, int *rowwindow_offset, int *vectorwindow_offset,
    int *edgeToColumn, int *edgeToRow,
    int *edgeList, half *values,
    int *sparseatob, int max_block, int num_nodes, long blockSize_h,
    int blockSize_w, int num_row_windows,
    int *b_rowwindow_offset_d, int *b_window_row_d, int *b_atomic_d, int part) {

  int tid = threadIdx.x;
  int winId = blockIdx.x; // each warp one window
  int group_offset = rowwindow_offset[winId];

  b_rowwindow_offset_d += group_offset;
  b_window_row_d += group_offset;
  b_atomic_d += group_offset;
  unsigned vector_start = vectorwindow_offset[winId];
  unsigned vector_end = vectorwindow_offset[min(winId + 1, num_row_windows)];
  unsigned num_vector = vector_end - vector_start;
  if (num_vector == 0) {
    return;
  }
  //根据part划分
  int block_num = (num_vector + blockSize_w - 1) / blockSize_w;
  int group_num = (block_num + part - 1) / part;
  if(group_num==1){
    b_rowwindow_offset_d[0] = num_vector;
    b_window_row_d[0] = winId;
    b_atomic_d[0] = 0;
  }
  else{
    for(int i=0; i<(group_num-1); i++)
    {      
      b_rowwindow_offset_d[0] = part*blockSize_w;
      b_window_row_d[0] = winId;
      b_atomic_d[0] = 1;

      b_rowwindow_offset_d++;
      b_window_row_d++;
      b_atomic_d++;
    }

    b_rowwindow_offset_d[0] = num_vector % (part*blockSize_w);
    b_window_row_d[0] = winId;
    b_atomic_d[0] = 1;
  }

  unsigned element_start = nodePointer[winId * blockSize_h];
  unsigned element_end =
      nodePointer[min(int(winId * blockSize_h + blockSize_h), num_nodes)];
  unsigned num_window_edges = element_end - element_start;
  if (num_window_edges == 0) {
    return;
  }
  //开始看每个非零元在block内的偏移了
//   auto tileid = tcblocktile_id + element_start;
  auto values_ = values + vector_start*blockSize_h;
  auto sparse_AToB = sparseatob + vector_start;
  for (unsigned e_index = element_start; e_index < element_end; e_index++) {
    unsigned col = edgeToColumn[e_index]; // new col
    unsigned tcblock_id = col / blockSize_w;
    unsigned row_local = edgeToRow[e_index] % blockSize_h;
    unsigned col_local = col % blockSize_w;

	//如果存在， 且元素在residue里，需要按每行residue偏移
	int residue = num_vector % blockSize_w;
	if(residue>0 & col>=(num_vector-residue)){
		values_[tcblock_id*blockSize_h*blockSize_w + row_local*residue + col_local] = __float2half(1.0);
	}else{
		values_[tcblock_id*blockSize_h*blockSize_w + row_local*blockSize_w + col_local] = __float2half(1.0);
	}
	
	sparse_AToB[tcblock_id * blockSize_w + col_local] = edgeList[e_index];
    // pos_ptr[tcblock_id]++;
  }
}

void generate_tcoffset_id_atob_cuda_fs_balance(int *nodePointer, int *rowwindow_offset, int *vectorwindow_offset,
                                    int *edgeToColumn, int *edgeToRow,
                                    int *edgeList, half *values, int *sparseatob,
                                    int max_block, int num_nodes,
                                    int blockSize_h, int blockSize_w,
                                    int num_row_windows,
                                    int *b_rowwindow_offset_d, int *b_window_row_d, int *b_atomic_d, int part) {
  int block_size = 1;
  int window_count = num_row_windows;
  generate_tcoffset_id_atob_fs_balance<<<window_count, block_size>>>(
      nodePointer, rowwindow_offset, vectorwindow_offset,
      edgeToColumn, edgeToRow, edgeList,
    values, sparseatob, max_block, num_nodes,
      blockSize_h, blockSize_w, num_row_windows,
    b_rowwindow_offset_d, b_window_row_d, b_atomic_d, part);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}

void padding_up_8(int *input, int size) {
  int threadsPerBlock = 256;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
  roundup_to_multiple_of_eight<<<blocksPerGrid, threadsPerBlock>>>(input, size);
}
void get_padding_tileid(int *ori_offset, uint8_t *ori_tileid,
                        int *padded_offset, uint8_t *padded_tileid, int size) {
  int threadsPerBlock = 256;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
  get_padding_tileid_kernel<<<blocksPerGrid, threadsPerBlock>>>(
      ori_offset, ori_tileid, padded_offset, padded_tileid, size);
}


void print_first_20(const thrust::device_vector<int>& seg, const thrust::device_vector<int>& el, const std::string& label) {
    std::cout << label << "前 20 个值:" << std::endl;
    std::cout << "Seg: ";
    thrust::host_vector<int> host_seg(seg.begin(), seg.begin() + 176);
    for (int i = 0; i < 176; i++) {
        std::cout << host_seg[i] << " ";
    }
    std::cout << std::endl;
    
    std::cout << "EL: ";
    thrust::host_vector<int> host_el(el.begin(), el.begin() + 176);
    for (int i = 0; i < 176; i++) {
        std::cout << host_el[i] << " ";
    }
    std::cout << std::endl;
}

/*main function*/
std::tuple<torch::Tensor, int, torch::Tensor>
seg_sort_dequ_fs(int *seg, int *edgeLists, int *nodepointer, int *edgetocol,
              int *edgetorow, int *blockpartition, int *block_num, int *vector_num,
              int *rowwindow_offset, int blockSize_h, int blockSize_w,
              int num_nodes, int num_edges, int rowwindow_num) {
	thrust::device_ptr<int> Seg = thrust::device_pointer_cast(seg);
	thrust::device_vector<int> deviceSeg(Seg, Seg + num_edges);
  hipFree(seg);

	thrust::device_ptr<int> EL = thrust::device_pointer_cast(edgeLists);
	thrust::device_vector<int> deviceEL(EL, EL + num_edges);
	auto begin = thrust::make_zip_iterator(
		thrust::make_tuple(deviceSeg.begin(), deviceEL.begin()));
	auto end = thrust::make_zip_iterator(
		thrust::make_tuple(deviceSeg.end(), deviceEL.end()));

	thrust::sort(thrust::device, begin, end);

	// thrust::device_ptr<int> Counts = thrust::device_pointer_cast(edgeLists);
	// thrust::device_vector<int> deviceCounts(Counts, Counts + num_edges);
	generate_edgetocolumn_cuda_fs(
		nodepointer, edgeLists, thrust::raw_pointer_cast(&deviceEL[0]),
		edgetocol,
		blockpartition, block_num, vector_num, blockSize_h, blockSize_w, num_nodes);

	thrust::device_ptr<int> blockpartition_ptr =
		thrust::device_pointer_cast(blockpartition);
	thrust::device_ptr<int> rowwindow_offset_ptr =
		thrust::device_pointer_cast(rowwindow_offset + 1);
	thrust::device_vector<int> blockpartition_vector(
		blockpartition_ptr, blockpartition_ptr + rowwindow_num);
  hipFree(blockpartition);
	thrust::inclusive_scan(blockpartition_vector.begin(),
							blockpartition_vector.end(), rowwindow_offset_ptr);
  
	auto options_gpu =
		torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
	auto options_gpu_unit8 =
		torch::TensorOptions().dtype(torch::kUInt8).device(torch::kCUDA);
	thrust::device_ptr<int> bnum_ptr = thrust::device_pointer_cast(block_num);
	thrust::host_vector<int> bnum_vector(bnum_ptr, bnum_ptr + 1);
	int block_counter = bnum_vector[0];

	thrust::device_ptr<int> vnum_ptr = thrust::device_pointer_cast(vector_num);
	thrust::host_vector<int> vnum_vector(vnum_ptr, vnum_ptr + 1);
	long vector_counter = vnum_vector[0];

  	//声明最终的数据结构
  auto values_tensor = torch::zeros({vector_counter*blockSize_h}, torch::kFloat16).to(torch::kCPU);
  auto sparse_AToX_index_tensor = torch::zeros({vector_counter}, torch::kInt32).to(torch::kCPU);

	auto values = reinterpret_cast<half *>(values_tensor.data<at::Half>());
	auto sparse_AToX_index = sparse_AToX_index_tensor.data<int>();

  half *values_d;
  int *sparse_AToX_index_d;

  hipMalloc(&values_d, (values_tensor.size(0)) * sizeof(half));
  hipMalloc(&sparse_AToX_index_d, (sparse_AToX_index_tensor.size(0)) * sizeof(int));

  hipMemcpy(values_d, values , (values_tensor.size(0)) * sizeof(half), hipMemcpyHostToDevice);
  // hipMemcpy(sparse_AToX_index_d, sparse_AToX_index , (sparse_AToX_index_tensor.size(0)) * sizeof(int), hipMemcpyHostToDevice);

	generate_tcoffset_id_atob_cuda_fs(
		nodepointer, rowwindow_offset, edgetocol, edgetorow, edgeLists,
		values_d, sparse_AToX_index_d, 1,
		num_nodes, blockSize_h, blockSize_w, rowwindow_num);

  hipMemcpy(values, values_d, vector_counter*blockSize_h * sizeof(half), hipMemcpyDeviceToHost);
  hipMemcpy(sparse_AToX_index, sparse_AToX_index_d, vector_counter * sizeof(int), hipMemcpyDeviceToHost);


    hipFree(values_d);
    hipFree(sparse_AToX_index_d);

	return std::make_tuple(
						sparse_AToX_index_tensor,
							block_counter,values_tensor);
}




/*main function*/
std::tuple<torch::Tensor, int, torch::Tensor>
seg_sort_dequ_fs_ori(int *seg, int *edgeLists, int *nodepointer, int *edgetocol,
              int *edgetorow, int *blockpartition, int *block_num, int *vector_num,
              int *rowwindow_offset, int blockSize_h, int blockSize_w,
              int num_nodes, int num_edges, int rowwindow_num) {
	thrust::device_ptr<int> Seg = thrust::device_pointer_cast(seg);
	thrust::device_vector<int> deviceSeg(Seg, Seg + num_edges);
  hipFree(seg);

	thrust::device_ptr<int> EL = thrust::device_pointer_cast(edgeLists);
	thrust::device_vector<int> deviceEL(EL, EL + num_edges);
	auto begin = thrust::make_zip_iterator(
		thrust::make_tuple(deviceSeg.begin(), deviceEL.begin()));
	auto end = thrust::make_zip_iterator(
		thrust::make_tuple(deviceSeg.end(), deviceEL.end()));

	thrust::sort(thrust::device, begin, end);

	// thrust::device_ptr<int> Counts = thrust::device_pointer_cast(edgeLists);
	// thrust::device_vector<int> deviceCounts(Counts, Counts + num_edges);
	generate_edgetocolumn_cuda_fs_ori(
		nodepointer, edgeLists, thrust::raw_pointer_cast(&deviceEL[0]),
		edgetocol,
		blockpartition, block_num, vector_num, blockSize_h, blockSize_w, num_nodes);

	thrust::device_ptr<int> blockpartition_ptr =
		thrust::device_pointer_cast(blockpartition);
	thrust::device_ptr<int> rowwindow_offset_ptr =
		thrust::device_pointer_cast(rowwindow_offset + 1);
	thrust::device_vector<int> blockpartition_vector(
		blockpartition_ptr, blockpartition_ptr + rowwindow_num);
  hipFree(blockpartition);
	thrust::inclusive_scan(blockpartition_vector.begin(),
							blockpartition_vector.end(), rowwindow_offset_ptr);
  
	auto options_gpu =
		torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
	auto options_gpu_unit8 =
		torch::TensorOptions().dtype(torch::kUInt8).device(torch::kCUDA);
	thrust::device_ptr<int> bnum_ptr = thrust::device_pointer_cast(block_num);
	thrust::host_vector<int> bnum_vector(bnum_ptr, bnum_ptr + 1);
	long block_counter = bnum_vector[0];

	thrust::device_ptr<int> vnum_ptr = thrust::device_pointer_cast(vector_num);
	thrust::host_vector<int> vnum_vector(vnum_ptr, vnum_ptr + 1);
	int vector_counter = vnum_vector[0];

  	//声明最终的数据结构
  auto values_tensor = torch::zeros({block_counter*blockSize_h*blockSize_w}, torch::kFloat16).to(torch::kCPU);
  auto sparse_AToX_index_tensor = torch::full({block_counter * blockSize_w}, -1, torch::kInt32).to(torch::kCPU);

	auto values = reinterpret_cast<half *>(values_tensor.data<at::Half>());
	auto sparse_AToX_index = sparse_AToX_index_tensor.data<int>();

  half *values_d;
  int *sparse_AToX_index_d;

  hipMalloc(&values_d, (values_tensor.size(0)) * sizeof(half));
  hipMalloc(&sparse_AToX_index_d, (sparse_AToX_index_tensor.size(0)) * sizeof(int));

  hipMemcpy(values_d, values , (values_tensor.size(0)) * sizeof(half), hipMemcpyHostToDevice);
  hipMemcpy(sparse_AToX_index_d, sparse_AToX_index , (sparse_AToX_index_tensor.size(0)) * sizeof(int), hipMemcpyHostToDevice);


	generate_tcoffset_id_atob_cuda_fs_ori(
		nodepointer, rowwindow_offset, edgetocol, edgetorow, edgeLists,
		values_d, sparse_AToX_index_d, 1,
		num_nodes, blockSize_h, blockSize_w, rowwindow_num);

  hipMemcpy(values, values_d, vector_counter*blockSize_h * sizeof(half), hipMemcpyDeviceToHost);
  hipMemcpy(sparse_AToX_index, sparse_AToX_index_d, vector_counter * sizeof(int), hipMemcpyDeviceToHost);


    hipFree(values_d);
    hipFree(sparse_AToX_index_d);

	return std::make_tuple(
						sparse_AToX_index_tensor,
							block_counter,values_tensor);
}



std::tuple<torch::Tensor, int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
seg_sort_dequ_fs_balance(int *seg, int *edgeLists, int *nodepointer, int *edgetocol,
              int *edgetorow, int *blockpartition, int *vectorPartition, int *block_num, int *vector_num,
              int *rowwindow_offset, int * vectorwindow_offset, 
              int blockSize_h, int blockSize_w,
              int num_nodes, int num_edges, int rowwindow_num, int part) {
	thrust::device_ptr<int> Seg = thrust::device_pointer_cast(seg);
	thrust::device_vector<int> deviceSeg(Seg, Seg + num_edges);
  hipFree(seg);

	thrust::device_ptr<int> EL = thrust::device_pointer_cast(edgeLists);
	thrust::device_vector<int> deviceEL(EL, EL + num_edges);
	auto begin = thrust::make_zip_iterator(
		thrust::make_tuple(deviceSeg.begin(), deviceEL.begin()));
	auto end = thrust::make_zip_iterator(
		thrust::make_tuple(deviceSeg.end(), deviceEL.end()));

	thrust::sort(thrust::device, begin, end);

  //确定每个window需要几个group,且返回后需要累加
  //确定每个window中的vector个数，且返回后不需要累加
	generate_edgetocolumn_cuda_fs_balance(
		nodepointer, edgeLists, thrust::raw_pointer_cast(&deviceEL[0]),
		edgetocol,
		blockpartition, vectorPartition, block_num, vector_num, blockSize_h, blockSize_w, num_nodes, part);

	thrust::device_ptr<int> blockpartition_ptr =
		thrust::device_pointer_cast(blockpartition);
	thrust::device_ptr<int> rowwindow_offset_ptr =
		thrust::device_pointer_cast(rowwindow_offset + 1);
	thrust::device_vector<int> blockpartition_vector(
		blockpartition_ptr, blockpartition_ptr + rowwindow_num);
  hipFree(blockpartition);
	thrust::inclusive_scan(blockpartition_vector.begin(),
							blockpartition_vector.end(), rowwindow_offset_ptr);

	thrust::device_ptr<int> vectorpartition_ptr =
		thrust::device_pointer_cast(vectorPartition);
	thrust::device_ptr<int> vectorwindow_offset_ptr =
		thrust::device_pointer_cast(vectorwindow_offset + 1);
	thrust::device_vector<int> vectorpartition_vector(
		vectorpartition_ptr, vectorpartition_ptr + rowwindow_num);
  hipFree(vectorPartition);
	thrust::inclusive_scan(vectorpartition_vector.begin(),
							vectorpartition_vector.end(), vectorwindow_offset_ptr);
  
	auto options_gpu =
		torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
	auto options_gpu_unit8 =
		torch::TensorOptions().dtype(torch::kUInt8).device(torch::kCUDA);
	thrust::device_ptr<int> bnum_ptr = thrust::device_pointer_cast(block_num);
	thrust::host_vector<int> bnum_vector(bnum_ptr, bnum_ptr + 1);
	int block_counter = bnum_vector[0];
  // printf("%d\n", block_counter);

	thrust::device_ptr<int> vnum_ptr = thrust::device_pointer_cast(vector_num);
	thrust::host_vector<int> vnum_vector(vnum_ptr, vnum_ptr + 1);
	long vector_counter = vnum_vector[0];

  	//声明最终的数据结构
  auto values_tensor = torch::zeros({vector_counter*blockSize_h}, torch::kFloat16).to(torch::kCPU);
  auto sparse_AToX_index_tensor = torch::zeros({vector_counter}, torch::kInt32).to(torch::kCPU);
  //根据block_counter确定b_rowwindow_offset_tensor， b_window_rowTensor， b_atomicTensor
  // auto b_rowwindow_offsetTensor = torch::zeros({block_counter+1}, torch::kInt32).to(torch::kCPU);
  auto b_window_rowTensor = torch::zeros({block_counter}, torch::kInt32).to(torch::kCPU);
  auto b_atomicTensor = torch::zeros({block_counter}, torch::kInt32).to(torch::kCPU);

	auto values = reinterpret_cast<half *>(values_tensor.data<at::Half>());
	auto sparse_AToX_index = sparse_AToX_index_tensor.data<int>();
	// auto b_rowwindow_offset = b_rowwindow_offsetTensor.data<int>();
	auto b_window_row = b_window_rowTensor.data<int>();
	auto b_atomic = b_atomicTensor.data<int>();

  half *values_d;
  int *sparse_AToX_index_d, *b_rowwindow_offset_d, *b_window_row_d, *b_atomic_d;

  hipMalloc(&values_d, (values_tensor.size(0)) * sizeof(half));
  hipMalloc(&sparse_AToX_index_d, (sparse_AToX_index_tensor.size(0)) * sizeof(int));
  hipMalloc(&b_rowwindow_offset_d, (block_counter) * sizeof(int));
  hipMalloc(&b_window_row_d, block_counter * sizeof(int));
  hipMalloc(&b_atomic_d, block_counter * sizeof(int));

  hipMemcpy(values_d, values , (values_tensor.size(0)) * sizeof(half), hipMemcpyHostToDevice);


	generate_tcoffset_id_atob_cuda_fs_balance(
		nodepointer, rowwindow_offset, vectorwindow_offset, edgetocol, edgetorow, edgeLists,
		values_d, sparse_AToX_index_d, 1,
		num_nodes, blockSize_h, blockSize_w, rowwindow_num,
    b_rowwindow_offset_d, b_window_row_d, b_atomic_d, part);

  hipMemcpy(values, values_d, vector_counter*blockSize_h * sizeof(half), hipMemcpyDeviceToHost);
  hipMemcpy(sparse_AToX_index, sparse_AToX_index_d, vector_counter * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b_window_row, b_window_row_d, block_counter * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b_atomic, b_atomic_d, block_counter * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(values_d);
  hipFree(sparse_AToX_index_d);
  hipFree(b_window_row_d);
  hipFree(b_atomic_d);

  //累加
  auto b_rowwindow_offset_outTensor = torch::zeros({block_counter+1}, torch::kInt32).to(torch::kCPU);
  auto b_rowwindow_offset_out = b_rowwindow_offset_outTensor.data<int>();
  int *b_rowwindow_offset_out_d;
  hipMalloc(&b_rowwindow_offset_out_d, (block_counter+1) * sizeof(int));
  hipMemset(b_rowwindow_offset_out_d, 0, (block_counter + 1) * sizeof(int));

	thrust::device_ptr<int> b_rowwindow_offset_d_ptr =
		thrust::device_pointer_cast(b_rowwindow_offset_d);
	thrust::device_ptr<int> b_rowwindow_offset_out_d_ptr =
		thrust::device_pointer_cast(b_rowwindow_offset_out_d + 1);
	thrust::device_vector<int> b_rowwindow_offset_d_vector(
		b_rowwindow_offset_d_ptr, b_rowwindow_offset_d_ptr + block_counter);
	thrust::inclusive_scan(b_rowwindow_offset_d_vector.begin(),
							b_rowwindow_offset_d_vector.end(), b_rowwindow_offset_out_d_ptr);
              
  hipMemcpy(b_rowwindow_offset_out, b_rowwindow_offset_out_d, (block_counter+1) * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(b_rowwindow_offset_d);
  hipFree(b_rowwindow_offset_out_d);


	return std::make_tuple(
						sparse_AToX_index_tensor,
							block_counter,values_tensor, b_rowwindow_offset_outTensor, b_window_rowTensor, b_atomicTensor);
}